/**
 * Copyright 2016 Andreas Schäfer
 *
 * Distributed under the Boost Software License, Version 1.0. (See accompanying
 * file LICENSE or copy at http://www.boost.org/LICENSE_1_0.txt)
 */

#include <boost/detail/lightweight_test.hpp>
#include <libflatarray/flat_array.hpp>
#include <map>

#include "test.hpp"

class ActiveElement
{
public:
    __host__
    __device__
    ActiveElement()
    {
        val += 100000;
    }

    __host__
    __device__
    ~ActiveElement()
    {
        val += 1000000;
    }

    inline bool operator==(ActiveElement other) const
    {
        return val == other.val;
    }

    int val;
};

class PassiveElement
{
public:
    inline bool operator==(PassiveElement other) const
    {
        return val == other.val;
    }

    int val;
};

class ConstructorDestructorTestCellActive
{
public:
    inline
    explicit ConstructorDestructorTestCellActive(double temperature=0.0, bool alive=false) :
        temperature(temperature),
        alive(alive)
    {}

    inline bool operator==(const ConstructorDestructorTestCellActive& other) const
    {
        return
            (temperature == other.temperature) &&
            (alive == other.alive) &&
            (element == other.element);
    }

    inline bool operator!=(const ConstructorDestructorTestCellActive& other) const
    {
        return !(*this == other);
    }

    double temperature;
    bool alive;
    ActiveElement element;
};

class ConstructorDestructorTestCellPassive
{
public:
    inline
    explicit ConstructorDestructorTestCellPassive(double temperature=0.0, bool alive=false) :
        temperature(temperature),
        alive(alive)
    {}

    inline bool operator==(const ConstructorDestructorTestCellPassive& other) const
    {
        return
            (temperature == other.temperature) &&
            (alive == other.alive) &&
            (element == other.element);
    }

    inline bool operator!=(const ConstructorDestructorTestCellPassive& other) const
    {
        return !(*this == other);
    }

    double temperature;
    bool alive;
    PassiveElement element;
};

LIBFLATARRAY_REGISTER_SOA(ConstructorDestructorTestCellActive,
                          ((double)(temperature))
                          ((bool)(alive))
                          ((ActiveElement)(element)) )

LIBFLATARRAY_REGISTER_SOA(ConstructorDestructorTestCellPassive,
                          ((double)(temperature))
                          ((bool)(alive))
                          ((PassiveElement)(element)) )

namespace LibFlatArray {

std::map<std::size_t, char*> allocation_cache;

/**
 * We fake allocation here to make sure our grids in the tests below
 * get the same pointers. We need this to be sure that we're working
 * on the same memory region with each.
 */
template<class T>
class fake_cuda_allocator
{
public:
    typedef ptrdiff_t difference_type;
    typedef T* pointer;
    typedef const T* const_pointer;
    typedef T& reference;
    typedef const T& const_reference;
    typedef T value_type;

    pointer allocate(std::size_t n, const void* = 0)
    {
        if (allocation_cache[n] != 0) {
            return allocation_cache[n];
        }


        pointer ret = 0;
        hipMalloc(&ret, n * sizeof(T));
        allocation_cache[n] = ret;
        return ret;
    }

    void deallocate(pointer p, std::size_t)
    {
        // intentionally left blank
    }

    void deallocate_all()
    {
        for (typename std::map<std::size_t, pointer>::iterator i = allocation_cache.begin(); i != allocation_cache.end(); ++i) {
            hipFree(i->second);
            i->second = 0;
        }
    }
};

ADD_TEST(TestCUDAConstructionDestruction)
{
    char *data = 0;
    {
        // prep device memory with consecutive numbers:
        soa_grid<ConstructorDestructorTestCellPassive, fake_cuda_allocator<char>, true> grid(20, 10, 5);
        data = grid.get_data();

        soa_grid<ConstructorDestructorTestCellPassive> buffer(20, 10, 5);
        for (int z = 0; z < 5; ++z) {
            for (int y = 0; y < 10; ++y) {
                for (int x = 0; x < 20; ++x) {
                    ConstructorDestructorTestCellPassive cell;
                    cell.element.val = x + y * 20 + z * 20 * 10;
                    buffer.set(x, y, z, cell);

                    cell = buffer.get(x, y, z);
                }
            }
        }
        hipMemcpy(grid.get_data(), buffer.get_data(), grid.byte_size(), hipMemcpyHostToDevice);

    }
    {
        // ensure c-tor was run by checking increment on all elements:
        soa_grid<ConstructorDestructorTestCellActive,  fake_cuda_allocator<char>, true> grid(20, 10, 5);
        BOOST_TEST(data == grid.get_data());

        soa_grid<ConstructorDestructorTestCellPassive> buffer(20, 10, 5);
        hipMemcpy(buffer.get_data(), grid.get_data(), grid.byte_size(), hipMemcpyDeviceToHost);
        for (int z = 0; z < 5; ++z) {
            for (int y = 0; y < 10; ++y) {
                for (int x = 0; x < 20; ++x) {
                    ConstructorDestructorTestCellPassive cell = buffer.get(x, y, z);
                    int expected = x + y * 20 + z * 20 * 10 + 100000;

                    BOOST_TEST(cell.element.val == expected);
                }
            }
        }
    }
    {
        // ensure d-tor was run by checking increment on all elements:
        soa_grid<ConstructorDestructorTestCellPassive> buffer(20, 10, 5);
        hipMemcpy(buffer.get_data(), data, buffer.byte_size(), hipMemcpyDeviceToHost);
        for (int z = 0; z < 5; ++z) {
            for (int y = 0; y < 10; ++y) {
                for (int x = 0; x < 20; ++x) {
                    ConstructorDestructorTestCellPassive cell = buffer.get(x, y, z);
                    int expected = x + y * 20 + z * 20 * 10 + 1100000;

                    BOOST_TEST(cell.element.val == expected);
                }
            }
        }
    }

    fake_cuda_allocator<char>().deallocate_all();
}

ADD_TEST(TestCUDAGetSetSingleElements)
{
    soa_grid<ConstructorDestructorTestCellPassive, cuda_allocator<char>, true> grid(40, 13, 8);

    for (int z = 0; z < 8; ++z) {
        for (int y = 0; y < 13; ++y) {
            for (int x = 0; x < 40; ++x) {
                ConstructorDestructorTestCellPassive cell;
                cell.element.val = 10000 + x + y * 40 + z * 40 * 13;
                grid.set(x, y, z, cell);
            }
        }
    }
}

}

int main(int argc, char **argv)
{
    return 0;
}
