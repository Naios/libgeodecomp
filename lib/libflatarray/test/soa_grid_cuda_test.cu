/**
 * Copyright 2016 Andreas Schäfer
 *
 * Distributed under the Boost Software License, Version 1.0. (See accompanying
 * file LICENSE or copy at http://www.boost.org/LICENSE_1_0.txt)
 */

#include <libflatarray/flat_array.hpp>
#include <map>

#include "test.hpp"

class ActiveElement
{
public:
    __host__
    __device__
    ActiveElement()
    {
        val += 100000;
    }

    __host__
    __device__
    ~ActiveElement()
    {
        val += 1000000;
    }

    inline bool operator==(ActiveElement other) const
    {
        return val == other.val;
    }

    int val;
};

class PassiveElement
{
public:
    inline bool operator==(PassiveElement other) const
    {
        return val == other.val;
    }

    int val;
};

class ConstructorDestructorTestCellActive
{
public:
    inline
    explicit ConstructorDestructorTestCellActive(double temperature=0.0, bool alive=false) :
        temperature(temperature),
        alive(alive)
    {}

    inline bool operator==(const ConstructorDestructorTestCellActive& other) const
    {
        return
            (temperature == other.temperature) &&
            (alive == other.alive) &&
            (element == other.element);
    }

    inline bool operator!=(const ConstructorDestructorTestCellActive& other) const
    {
        return !(*this == other);
    }

    double temperature;
    bool alive;
    ActiveElement element;
};

class ConstructorDestructorTestCellPassive
{
public:
    inline
    explicit ConstructorDestructorTestCellPassive(double temperature=0.0, bool alive=false) :
        temperature(temperature),
        alive(alive)
    {}

    inline bool operator==(const ConstructorDestructorTestCellPassive& other) const
    {
        return
            (temperature == other.temperature) &&
            (alive == other.alive) &&
            (element == other.element);
    }

    inline bool operator!=(const ConstructorDestructorTestCellPassive& other) const
    {
        return !(*this == other);
    }

    double temperature;
    bool alive;
    PassiveElement element;
};

class CellWithArrayMember
{
public:
    __host__
    __device__
    inline
    explicit CellWithArrayMember(int j = 0) :
        j(j)
    {
        i[0] = j + 1;
        i[1] = j + 2;
        i[2] = j + 3;

        x[0] = j + 0.4;
        x[1] = j + 0.5;
    }

    __host__
    __device__
    inline
    CellWithArrayMember(int newI[3], double newX[2], int j) :
        j(j)
    {
        i[0] = newI[0];
        i[1] = newI[1];
        i[1] = newI[2];

        x[0] = newX[0];
        x[1] = newX[1];
    }

    int i[3];
    int j;
    double x[2];
};

class CellWithActiveArrayMember
{
public:
    __host__
    __device__
    inline
    explicit CellWithActiveArrayMember(int j = 0) :
        j(j)
    {
        i[0] = j + 1;
        i[1] = j + 2;
        i[2] = j + 3;
    }

    int i[3];
    int j;
    ActiveElement elements[2];
};

class CellWithPassiveArrayMember
{
public:
    __host__
    __device__
    inline
    explicit CellWithPassiveArrayMember(int j = 0) :
        j(j)
    {
        i[0] = j + 1;
        i[1] = j + 2;
        i[2] = j + 3;
    }

    int i[3];
    int j;
    PassiveElement elements[2];
};

LIBFLATARRAY_REGISTER_SOA(ConstructorDestructorTestCellActive,
                          ((double)(temperature))
                          ((ActiveElement)(element))
                          ((bool)(alive)) )

LIBFLATARRAY_REGISTER_SOA(ConstructorDestructorTestCellPassive,
                          ((double)(temperature))
                          ((PassiveElement)(element))
                          ((bool)(alive)) )

LIBFLATARRAY_REGISTER_SOA(CellWithArrayMember,
                          ((int)(i)(3))
                          ((int)(j))
                          ((double)(x)(2)) )

LIBFLATARRAY_REGISTER_SOA(CellWithActiveArrayMember,
                          ((int)(i)(3))
                          ((int)(j))
                          ((ActiveElement)(elements)(2)) )

LIBFLATARRAY_REGISTER_SOA(CellWithPassiveArrayMember,
                          ((int)(i)(3))
                          ((int)(j))
                          ((PassiveElement)(elements)(2)) )

namespace LibFlatArray {

std::map<std::size_t, char*> allocation_cache;

/**
 * We fake allocation here to make sure our grids in the tests below
 * get the same pointers. We need this to be sure that we're working
 * on the same memory region with each.
 */
template<class T>
class fake_cuda_allocator
{
public:
    typedef ptrdiff_t difference_type;
    typedef T* pointer;
    typedef const T* const_pointer;
    typedef T& reference;
    typedef const T& const_reference;
    typedef T value_type;

    pointer allocate(std::size_t n, const void* = 0)
    {
        if (allocation_cache[n] != 0) {
            return allocation_cache[n];
        }


        pointer ret = 0;
        hipMalloc(&ret, n * sizeof(T));
        allocation_cache[n] = ret;
        return ret;
    }

    void deallocate(pointer p, std::size_t)
    {
        // intentionally left blank
    }

    void deallocate_all()
    {
        for (typename std::map<std::size_t, pointer>::iterator i = allocation_cache.begin(); i != allocation_cache.end(); ++i) {
            hipFree(i->second);
            i->second = 0;
        }
    }
};

ADD_TEST(TestCUDAConstructionDestruction)
{
    char *data = 0;
    {
        // prep device memory with consecutive numbers:
        soa_grid<ConstructorDestructorTestCellPassive, fake_cuda_allocator<char>, true> device_grid(20, 10, 5);
        data = device_grid.get_data();

        soa_grid<ConstructorDestructorTestCellPassive> host_grid(20, 10, 5);
        for (int z = 0; z < 5; ++z) {
            for (int y = 0; y < 10; ++y) {
                for (int x = 0; x < 20; ++x) {
                    ConstructorDestructorTestCellPassive cell((x + 1) * (y + 1), true);
                    cell.element.val = x + y * 20 + z * 20 * 10;
                    host_grid.set(x, y, z, cell);

                    cell = host_grid.get(x, y, z);
                }
            }
        }
        hipMemcpy(device_grid.get_data(), host_grid.get_data(), device_grid.byte_size(), hipMemcpyHostToDevice);

    }
    {
        // ensure c-tor was run by checking increment on all elements:
        soa_grid<ConstructorDestructorTestCellActive,  fake_cuda_allocator<char>, true> device_grid(20, 10, 5);
        BOOST_TEST(data == device_grid.get_data());

        soa_grid<ConstructorDestructorTestCellPassive> host_grid(20, 10, 5);
        hipMemcpy(host_grid.get_data(), device_grid.get_data(), device_grid.byte_size(), hipMemcpyDeviceToHost);
        for (int z = 0; z < 5; ++z) {
            for (int y = 0; y < 10; ++y) {
                for (int x = 0; x < 20; ++x) {
                    ConstructorDestructorTestCellPassive cell = host_grid.get(x, y, z);
                    int expected = x + y * 20 + z * 20 * 10 + 100000;

                    BOOST_TEST(cell.element.val == expected);
                    BOOST_TEST(cell.temperature == 0);
                    BOOST_TEST(cell.alive == false);
                }
            }
        }
    }
    {
        // ensure d-tor was run by checking increment on all elements:
        soa_grid<ConstructorDestructorTestCellPassive> host_grid(20, 10, 5);
        hipMemcpy(host_grid.get_data(), data, host_grid.byte_size(), hipMemcpyDeviceToHost);
        for (int z = 0; z < 5; ++z) {
            for (int y = 0; y < 10; ++y) {
                for (int x = 0; x < 20; ++x) {
                    ConstructorDestructorTestCellPassive cell = host_grid.get(x, y, z);
                    int expected = x + y * 20 + z * 20 * 10 + 1100000;

                    BOOST_TEST(cell.element.val == expected);
                    BOOST_TEST(cell.temperature == 0);
                    BOOST_TEST(cell.alive == false);
                }
            }
        }
    }

    fake_cuda_allocator<char>().deallocate_all();
}

ADD_TEST(TestCUDAGetSetSingleElements)
{
    soa_grid<ConstructorDestructorTestCellPassive, cuda_allocator<char>, true> device_grid(40, 13, 8);

    for (int z = 0; z < 8; ++z) {
        for (int y = 0; y < 13; ++y) {
            for (int x = 0; x < 40; ++x) {
                ConstructorDestructorTestCellPassive cell((x + 2) * (y + 2), true);
                cell.element.val = 10000 + x + y * 40 + z * 40 * 13;
                device_grid.set(x, y, z, cell);
            }
        }
    }

    for (int z = 0; z < 8; ++z) {
        for (int y = 0; y < 13; ++y) {
            for (int x = 0; x < 40; ++x) {
                ConstructorDestructorTestCellPassive cell = device_grid.get(x, y, z);

                int expected = 10000 + x + y * 40 + z * 40 * 13;
                BOOST_TEST(cell.element.val == expected);
                BOOST_TEST(cell.temperature == ((x + 2) * (y + 2)));
                BOOST_TEST(cell.alive       == true);
            }
        }
    }
}

ADD_TEST(TestCUDAGetSetMultipleElements)
{
    soa_grid<ConstructorDestructorTestCellPassive, cuda_allocator<char>, true> device_grid(35, 25, 15);

    for (int z = 0; z < 15; ++z) {
        for (int y = 0; y < 25; ++y) {
            std::vector<ConstructorDestructorTestCellPassive> cells(35);
            for (int x = 0; x < 35; ++x) {
                cells[x].alive = x % 2;
                cells[x].temperature = x * y * z;
                cells[x].element.val = 20000 + x + y * 35 + z * 35 * 25;
            }

            device_grid.set(0, y, z, cells.data(), 35);
        }
    }

    for (int z = 0; z < 15; ++z) {
        for (int y = 0; y < 25; ++y) {
            std::vector<ConstructorDestructorTestCellPassive> cells(35);
            device_grid.get(0, y, z, cells.data(), 35);

            for (int x = 0; x < 35; ++x) {
                int expected = 20000 + x + y * 35 + z * 35 * 25;

                BOOST_TEST(cells[x].element.val == expected);
                BOOST_TEST(cells[x].alive == (x % 2));
                BOOST_TEST(cells[x].temperature == (x * y * z));
            }
        }
    }
}

ADD_TEST(TestCUDALoadSaveElements)
{
    soa_grid<ConstructorDestructorTestCellPassive> host_grid(21, 10, 9);
    for (int z = 0; z < 9; ++z) {
        for (int y = 0; y < 10; ++y) {
            for (int x = 0; x < 21; ++x) {
                ConstructorDestructorTestCellPassive cell;
                cell.alive = ((x % 3) == 0);
                cell.temperature = x * y * z * -1;
                cell.element.val = 30000 + x + y * 21 + z * 21 * 10;
                host_grid.set(x, y, z, cell);
            }
        }
    }

    std::vector<char> buffer(10 * aggregated_member_size<ConstructorDestructorTestCellPassive>::VALUE);
    host_grid.save(11, 9, 8, buffer.data(), 10);

    soa_grid<ConstructorDestructorTestCellPassive, cuda_allocator<char>, true> device_grid(31, 20, 19);
    device_grid.load(21, 19, 18, buffer.data(), 10);

    for (int i = 0; i < 20; ++i) {
        ConstructorDestructorTestCellPassive cell;
        cell.alive = i % 4;
        cell.temperature = 4711 + i;
        cell.element.val = 100 * i;
        device_grid.set(i + 1, 5, 6, cell);
    }

    buffer.resize(20 * aggregated_member_size<ConstructorDestructorTestCellPassive>::VALUE);
    device_grid.save(1, 5, 6, buffer.data(), 20);

    // very load:
    soa_grid<ConstructorDestructorTestCellPassive> host_grid2(31, 20, 19);
    hipMemcpy(host_grid2.get_data(), device_grid.get_data(), device_grid.byte_size(), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i) {
        ConstructorDestructorTestCellPassive cell = host_grid2.get(21 + i, 19, 18);

        bool expectedAlive = (((i + 11) % 3) == 0);
        double expectedTemperature = (11 + i) * 9 * 8 * -1;
        int expectedVal = 30000 + (11 + i) + 9 * 21 + 8 * 21 * 10;

        BOOST_TEST(cell.alive == expectedAlive);
        BOOST_TEST(cell.temperature == expectedTemperature);
        BOOST_TEST(cell.element.val == expectedVal);
    }

    // verify save:
    double *temperature = (double*)(buffer.data() +  0 * 20);
    int *val            = (int*)   (buffer.data() +  8 * 20);
    bool *alive         = (bool*)  (buffer.data() + 12 * 20);

    for (int i = 0; i < 20; ++i) {
        bool expectedAlive = i % 4;
        double expectedTemperature = 4711 + i;
        int expectedVal = i * 100;

        BOOST_TEST(expectedAlive       == alive[i]);
        BOOST_TEST(expectedTemperature == temperature[i]);
        BOOST_TEST(expectedVal         == val[i]);
    }

    // sanity check:
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        std::cerr << "ERROR: " << hipGetErrorString(error) << "\n";
        throw std::runtime_error("CUDA error");
    }
}

ADD_TEST(TestCUDAArrayMembersGetSet)
{
    // test set/get single elements:
    soa_grid<CellWithArrayMember, cuda_allocator<char>, true> device_grid(12, 23, 34);

    for (int z = 0; z < 34; ++z) {
        for (int y = 0; y < 23; ++y) {
            for (int x = 0; x < 12; ++x) {
                CellWithArrayMember cell;
                cell.i[0] = x;
                cell.i[1] = y;
                cell.i[2] = z;
                cell.j    = x * y * z;
                cell.x[0] = x + y + 0.1;
                cell.x[1] = y + z + 0.2;

                device_grid.set(x, y, z, cell);
            }
        }
    }

    for (int z = 0; z < 34; ++z) {
        for (int y = 0; y < 23; ++y) {
            for (int x = 0; x < 12; ++x) {
                int expectedCellI0 = x;
                int expectedCellI1 = y;
                int expectedCellI2 = z;
                int expectedCellJ  = x * y * z;
                double expectedCellX0 = x + y + 0.1;
                double expectedCellX1 = y + z + 0.2;

                CellWithArrayMember cell = device_grid.get(x, y, z);

                BOOST_TEST(expectedCellI0 == cell.i[0]);
                BOOST_TEST(expectedCellI1 == cell.i[1]);
                BOOST_TEST(expectedCellI2 == cell.i[2]);

                BOOST_TEST(expectedCellJ  == cell.j);

                BOOST_TEST(expectedCellX0 == cell.x[0]);
                BOOST_TEST(expectedCellX1 == cell.x[1]);
            }
        }
    }
}

ADD_TEST(TestCUDAArrayMembersGetSetMultiple)
{
    // test set/get single elements:
    soa_grid<CellWithArrayMember, cuda_allocator<char>, true> device_grid(40, 23, 34);

    for (int z = 0; z < 34; ++z) {
        for (int y = 0; y < 23; ++y) {
            CellWithArrayMember cells[40];
            for (int x = 0; x < 40; ++x) {
                cells[x].i[0] = x;
                cells[x].i[1] = y;
                cells[x].i[2] = z;
                cells[x].j    = x * y * z;
                cells[x].x[0] = x + y + 0.1;
                cells[x].x[1] = y + z + 0.2;
            }

            device_grid.set(0, y, z, cells, 40);
        }
    }

    for (int z = 0; z < 34; ++z) {
        for (int y = 0; y < 23; ++y) {
            CellWithArrayMember cells[40];
            device_grid.get(0, y, z, cells, 40);

            for (int x = 0; x < 40; ++x) {
                int expectedCellI0 = x;
                int expectedCellI1 = y;
                int expectedCellI2 = z;
                int expectedCellJ  = x * y * z;
                double expectedCellX0 = x + y + 0.1;
                double expectedCellX1 = y + z + 0.2;

                BOOST_TEST(expectedCellI0 == cells[x].i[0]);
                BOOST_TEST(expectedCellI1 == cells[x].i[1]);
                BOOST_TEST(expectedCellI2 == cells[x].i[2]);

                BOOST_TEST(expectedCellJ  == cells[x].j);

                BOOST_TEST(expectedCellX0 == cells[x].x[0]);
                BOOST_TEST(expectedCellX1 == cells[x].x[1]);
            }
        }
    }
}

ADD_TEST(TestCUDAArrayMembersConstructDestruct)
{
    char *data = 0;
    {
        // prep device memory with consecutive numbers:
        soa_grid<CellWithPassiveArrayMember, fake_cuda_allocator<char>, true> device_grid(8, 9, 13);
        data = device_grid.get_data();

        soa_grid<CellWithPassiveArrayMember> host_grid(8, 9, 13);
        for (int z = 0; z < 13; ++z) {
            for (int y = 0; y < 9; ++y) {
                for (int x = 0; x < 8; ++x) {
                    CellWithPassiveArrayMember cell((x + 1) * (y + 1));
                    cell.elements[0].val = 40000 + x + y * 8 + z * 8 * 9;
                    cell.elements[1].val = 50000 + x + y * 8 + z * 8 * 9;
                    host_grid.set(x, y, z, cell);

                    cell = host_grid.get(x, y, z);
                }
            }
        }
        hipMemcpy(device_grid.get_data(), host_grid.get_data(), device_grid.byte_size(), hipMemcpyHostToDevice);

    }
    {
        // ensure c-tor was run by checking increment on all elements:
        soa_grid<CellWithActiveArrayMember,  fake_cuda_allocator<char>, true> device_grid(8, 9, 13);
        BOOST_TEST(data == device_grid.get_data());

        soa_grid<CellWithPassiveArrayMember> host_grid(8, 9, 13);
        hipMemcpy(host_grid.get_data(), device_grid.get_data(), device_grid.byte_size(), hipMemcpyDeviceToHost);
        for (int z = 0; z < 13; ++z) {
            for (int y = 0; y < 9; ++y) {
                for (int x = 0; x < 8; ++x) {
                    CellWithPassiveArrayMember cell = host_grid.get(x, y, z);
                    int expected0 = 40000 + x + y * 8 + z * 8 * 9 + 100000;
                    int expected1 = 50000 + x + y * 8 + z * 8 * 9 + 100000;

                    BOOST_TEST(cell.elements[0].val == expected0);
                    BOOST_TEST(cell.elements[1].val == expected1);

                    BOOST_TEST(cell.i[0] == 0);
                    BOOST_TEST(cell.i[1] == 0);
                    BOOST_TEST(cell.i[2] == 0);
                }
            }
        }
    }
    {
        // ensure d-tor was run by checking increment on all elements:
        soa_grid<CellWithPassiveArrayMember> host_grid(8, 9, 13);
        hipMemcpy(host_grid.get_data(), data, host_grid.byte_size(), hipMemcpyDeviceToHost);
        for (int z = 0; z < 13; ++z) {
            for (int y = 0; y < 9; ++y) {
                for (int x = 0; x < 8; ++x) {
                    CellWithPassiveArrayMember cell = host_grid.get(x, y, z);
                    int expected0 = 40000 + x + y * 8 + z * 8 * 9 + 1100000;
                    int expected1 = 50000 + x + y * 8 + z * 8 * 9 + 1100000;

                    BOOST_TEST(cell.elements[0].val == expected0);
                    BOOST_TEST(cell.elements[1].val == expected1);

                    BOOST_TEST(cell.i[0] == 0);
                    BOOST_TEST(cell.i[1] == 0);
                    BOOST_TEST(cell.i[2] == 0);
                }
            }
        }
    }

    fake_cuda_allocator<char>().deallocate_all();
}

ADD_TEST(TestCUDAArrayMembersLoadSave)
{
    soa_grid<CellWithPassiveArrayMember, cuda_allocator<char>, true> device_grid(45, 35, 25);
    for (int z = 0; z < 25; ++z) {
        for (int y = 0; y < 35; ++y) {
            for (int x = 0; x < 45; ++x) {
                CellWithPassiveArrayMember cell;
                cell.i[0] = x;
                cell.i[1] = y;
                cell.i[2] = z;
                cell.j = x * y * z;
                cell.elements[0].val = 4711 + x * y;
                cell.elements[1].val =  666 + y * z;

                device_grid.set(x, y, z, cell);
            }
        }
    }

    std::vector<char> buffer(aggregated_member_size<CellWithPassiveArrayMember>::VALUE * 33);
    device_grid.save(12, 34, 24, buffer.data(), 33);

    soa_grid<CellWithPassiveArrayMember, cuda_allocator<char>, true> device_grid2(35, 20, 5);
    device_grid2.load(2, 19, 4, buffer.data(), 33);

    for (int x = 0; x < 33; ++x) {
        CellWithPassiveArrayMember cell = device_grid2.get(x + 2, 19, 4);

        int expectedI0 = x + 12;
        int expectedI1 = 34;
        int expectedI2 = 24;

        int expectedJ = (x + 12) * 34 * 24;

        int expectedElements0 = 4711 + (x + 12) * 34;
        int expectedElements1 =  666 + 34 * 24;

        BOOST_TEST(cell.i[0] == expectedI0);
        BOOST_TEST(cell.i[1] == expectedI1);
        BOOST_TEST(cell.i[2] == expectedI2);

        BOOST_TEST(cell.j == expectedJ);

        BOOST_TEST(cell.elements[0].val == expectedElements0);
        BOOST_TEST(cell.elements[1].val == expectedElements1);
    }
}

}

int main(int argc, char **argv)
{
    return 0;
}
