#include <libgeodecomp/misc/chronometer.h>
#include <libgeodecomp/testbed/reversetimemigration/cudastepperlib.h>
#include <libgeodecomp/testbed/reversetimemigration/cudastepper.h>

using namespace LibGeoDecomp;
using namespace HiParSimulator;

class RTMCell
{
public:
    typedef Topologies::Cube<3>::Topology Topology;
    static int flops()
    {
        return 1;
    }

};

void CudaStepperLib::doit(const int& deviceID)
{
    std::cout << "setting device " << deviceID << "\n";
    hipSetDevice(deviceID);
    CUDAStepper<RTMCell> stepper;

    long long timeStart = Chronometer::timeUSec();

    int repeats = 100;
    for (int i = 0; i < repeats; ++i) {
        stepper.step();
    }

    hipDeviceSynchronize();
    long long timeEnd = Chronometer::timeUSec();
    double updates = 1.0 * repeats * DIM_Z * GRID_DIM_X * GRID_DIM_Y * BLOCK_DIM_X * BLOCK_DIM_Y;
    double time = (timeEnd - timeStart) * 0.0000001;
    double glups = updates / time * 0.0000000001;
    double gflops = glups * RTMCell::flops();

    std::cout << "GLUPS:  " << glups << "\n";
    std::cout << "GFLOPS: " << gflops << "\n";

    CUDAStepper<RTMCell>::checkForCUDAError();


}
