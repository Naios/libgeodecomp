#include <libgeodecomp/misc/chronometer.h>
#include <libgeodecomp/testbed/reversetimemigration/cudastepperlib.h>
#include <libgeodecomp/testbed/reversetimemigration/cudastepper.h>

using namespace LibGeoDecomp;
using namespace HiParSimulator;

class RTMCell
{
public:
    class API : public APITraits::HasCubeTopology<3>
    {};

    static int flops()
    {
        return 50;
    }

};

void CudaStepperLib::doit(const int& deviceID)
{
    std::cout << "setting device " << deviceID << "\n";
    hipSetDevice(deviceID);
    CUDAStepper<RTMCell> stepper;

    long long timeStart = ScopedTimer::timeUSec();

    int repeats = 100;
    for (int i = 0; i < repeats; ++i) {
        stepper.step();
    }

    hipDeviceSynchronize();
    long long timeEnd = ScopedTimer::timeUSec();
    double updates = 1.0 * repeats * (DIM_Z - 4)* GRID_DIM_X * GRID_DIM_Y * BLOCK_DIM_X * BLOCK_DIM_Y;
    double time = (timeEnd - timeStart) * 0.0000001;
    double glups = updates / time * 0.0000000001;
    double gflops = glups * RTMCell::flops();

    std::cout << "GLUPS:  " << glups << "\n";
    std::cout << "GFLOPS: " << gflops << "\n";

    CUDAStepper<RTMCell>::checkForCUDAError();


}
